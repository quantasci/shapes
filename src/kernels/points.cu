#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
// FLUIDS v5.0 - SPH Fluid Simulator for CPU and GPU
// Copyright (C) 2012-2013, 2021. Rama Hoetzlein, http://fluids3.com
//-----------------------------------------------------------------------------


#define CUDA_KERNEL
#include "points.cuh"

#include "hip/hip_fp16.h"			// for 16-bit height fields
#include "cutil_math.h"			// cutil32.lib
#include <string.h>
#include <assert.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "datax.h"

__constant__ FParams_t	FParams;		
__constant__ cuDataX	FPnts;
__constant__ cuDataX	FPntTmp;
__constant__ cuDataX	FAccel;

#define SCAN_BLOCKSIZE		512

#define maxf(a,b)  (a>b ? a : b)

__device__ int getGridCell ( float3 pos, int3& gc )
{	
	gc.x = (int)( (pos.x - FParams.gridMin.x) * FParams.gridDelta.x);			// Cell in which particle is located
	gc.y = (int)( (pos.y - FParams.gridMin.y) * FParams.gridDelta.y);
	gc.z = (int)( (pos.z - FParams.gridMin.z) * FParams.gridDelta.z);		
	return ( (gc.x >= 1 && gc.x <= FParams.gridScanMax.x && gc.y >= 1 && gc.y <= FParams.gridScanMax.y && gc.z >= 1 && gc.z <= FParams.gridScanMax.z) ? (int) ((gc.y*FParams.gridRes.z + gc.z)*FParams.gridRes.x + gc.x) : GRID_UNDEF );
}


extern "C" __global__ void insertParticles ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;
	//if ( FPnts.bufI(FGCELL)[i] == GRID_UNDEF) return;		//--- to persist remove particles, overall approach must change
		
	register int3		gc;	
	
	int gs = getGridCell ( FPnts.bufF3(FPOS)[i], gc );	
	FPnts.bufI(FGCELL)[i] = gs;
	if ( gs != GRID_UNDEF ) {	
		FPnts.bufI(FGNDX)[i] = atomicAdd ( &FAccel.bufI(AGRIDCNT)[ gs ], 1 );		// Grid counts.
	} 
}


// debugAccess - very useful function to show all GPU pointers
__device__ void debugAccess ()
{
	printf ( "--- gpu bufs\n" );
	for (int i=0; i < 8; i++)
		printf ( "%d: %012llx   %012llx\n", i, FPnts.bufI(i), FPntTmp.bufI(i) );	
}


extern "C" __global__ void countingSortFull ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;		// particle index				
	if ( i >= pnum ) return;

	// This algorithm is O(2NK) in space, O(N/P) time, where K=sizeof(Fluid)
	// Copy particle from original, unsorted buffer (msortbuf),
	// into sorted memory location on device (mpos/mvel). 
	
	// **NOTE** We cannot use shared memory for temporary storage since this is a 
	// global reordering and there is no synchronization across blocks. 

	int icell = FPntTmp.bufI(FGCELL) [ i ];	
	if ( icell == GRID_UNDEF ) return;

	// Determine the sort_ndx; location of the particle after sort			
	 int indx = FPntTmp.bufI(FGNDX)[i];
	int sort_ndx = FAccel.bufI(AGRIDOFF) [ icell ] + indx ;	// global_ndx = grid_cell_offet + particle_offset	
		
	// Transfer data to sort location		
	FPnts.bufF3(FPOS)[sort_ndx] =		FPntTmp.bufF3(FPOS)[i];
	FPnts.bufF3(FVEL)[sort_ndx] =		FPntTmp.bufF3(FVEL)[i];		
	FPnts.bufI (FCLR)[sort_ndx] =		FPntTmp.bufI(FCLR) [i];
	
	FPnts.bufI(FGCELL)[sort_ndx] = icell;									// accel buffers
	FPnts.bufI(FGNDX)[sort_ndx] = indx;

	if (FPnts.bufF3(FVEVAL) != 0x0) {											// advect buffers
		FPnts.bufF3(FVEVAL)[sort_ndx] =		FPntTmp.bufF3(FVEVAL) [i];			
		FPnts.bufF3(FFORCE)[sort_ndx] =		FPntTmp.bufF3(FFORCE) [i];	
	}
	if (FPnts.bufF3(FPRESS) != 0x0) {
		FPnts.bufF (FPRESS)[sort_ndx] =		FPntTmp.bufF(FPRESS) [i];			// sph buffers
	} 

	FAccel.bufI (AGRID) [ sort_ndx ] =	sort_ndx;			// full sort, grid indexing becomes identity					
} 

extern "C" __device__ float contributePressure ( int i, float3 p, int cell )
{			
	if ( FAccel.bufI(AGRIDCNT)[cell] == 0 ) return 0.0;

	float3 dist;
	float dsq, sum = 0.0;
	register float d2 = FParams.sim_scale * FParams.sim_scale;
	register float r2 = FParams.r2 / d2;
	
	int clast = FAccel.bufI(AGRIDOFF)[cell] + FAccel.bufI(AGRIDCNT)[cell];

	for ( int cndx = FAccel.bufI(AGRIDOFF)[cell]; cndx < clast; cndx++ ) {
		int pndx = FAccel.bufI(AGRID) [cndx];
		dist = p - FPnts.bufF3(FPOS) [pndx];
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
		if ( dsq < r2 && dsq > 0.0) {
			dsq = (r2 - dsq)*d2;
			sum += dsq * dsq * dsq;				
		} 
	}	
	return sum;
}
			
extern "C" __global__ void computePressure ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	// Get search cell	
	uint gc = FPnts.bufI(FGCELL) [i];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	gc -= (1*FParams.gridRes.z + 1)*FParams.gridRes.x + 1;

	float3 dist;
	float dsq, sum = 0.0;
	register int cell;

	// Sum Pressures
	float3 pos = FPnts.bufF3(FPOS) [i];

	for (int c=0; c < FParams.gridAdjCnt; c++) {
		cell = gc + FParams.gridAdj[c];
		int clast = FAccel.bufI(AGRIDOFF)[cell] + FAccel.bufI(AGRIDCNT)[cell];
		for ( int cndx = FAccel.bufI(AGRIDOFF)[cell]; cndx < clast; cndx++ ) {			
			dist = pos - FPnts.bufF3(FPOS) [ FAccel.bufI(AGRID)[cndx] ];
			dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
			if ( dsq < FParams.rd2 && dsq > 0.0) {
				dsq = (FParams.rd2 - dsq) * FParams.d2;
				sum += dsq * dsq * dsq;				
			} 
		}			
		//--- not used (function call is slow, uses too many registers)
		// sum += contributePressure ( i, pos, gc + FParams.gridAdj[c] );
	}
	__syncthreads();
		
	// Compute Density & Pressure
	sum = sum * FParams.pmass * FParams.poly6kern;
	if ( sum == 0.0 ) sum = FParams.prest_dens;
	FPnts.bufF(FPRESS) [ i ] = sum;
}

extern "C" __device__ float3 contributeForce ( int i, float3 ipos, float3 iveleval, float di, float pi, int cell)
{			
	if ( FAccel.bufI(AGRIDCNT)[cell] == 0 ) return make_float3(0,0,0);	

	float dsq, c, pterm;	
	float3 dist, force = make_float3(0,0,0);
	float pj;
	int j;

	int clast = FAccel.bufI(AGRIDOFF)[cell] + FAccel.bufI(AGRIDCNT)[cell];

	for ( int cndx = FAccel.bufI(AGRIDOFF)[cell]; cndx < clast; cndx++ ) {
		
		j = FAccel.bufI(AGRID)[ cndx ];				
		dist = ( ipos - FPnts.bufF3(FPOS)[ j ] );		// dist in cm
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);

		if ( dsq < FParams.rd2 && dsq > 0) {			
			dsq = sqrt(dsq * FParams.d2);
			c = ( FParams.psmoothradius - dsq ); 
			pj = (FPnts.bufF(FPRESS)[j] - FParams.prest_dens ) * FParams.pintstiff;
			pterm = FParams.sim_scale * -0.5f * c * FParams.spikykern * ( pi + pj ) / dsq;			
			force += (  FParams.iterm * pterm * dist + FParams.vterm * ( FPnts.bufF3(FVEVAL)[ j ] - iveleval )) * c / (di * FPnts.bufF(FPRESS)[j]);
		}	
	}
	return force;
}

extern "C" __global__ void computeForce ( int pnum)
{			
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	// Get search cell	
	uint gc = FPnts.bufI(FGCELL)[ i ];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	gc -= (1*FParams.gridRes.z + 1)*FParams.gridRes.x + 1;

	// Sum Pressures	
	register int cell, c, j, cndx;
	register float3 force, dist;	
	register float pterm, dsq;
	float pi, pj;

	force = make_float3(0,0,0);			

	for ( c=0; c < FParams.gridAdjCnt; c++) {
		cell = gc + FParams.gridAdj[c];		

		for ( cndx = FAccel.bufI(AGRIDOFF)[cell]; cndx < FAccel.bufI(AGRIDOFF)[cell] + FAccel.bufI(AGRIDCNT)[cell]; cndx++ ) {
			j = FAccel.bufI(AGRID)[ cndx ];				
			dist = ( FPnts.bufF3(FPOS)[i] - FPnts.bufF3(FPOS)[ j ] );		// dist in cm
			dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
			if ( dsq < FParams.rd2 && dsq > 0) {			
				dsq = sqrt(dsq * FParams.d2);			
				pterm = FParams.sim_scale * -0.5f * (FParams.psmoothradius-dsq) * FParams.spikykern * FParams.pintstiff * ( FPnts.bufF(FPRESS)[i] + FPnts.bufF(FPRESS)[j] - 2*FParams.prest_dens ) / dsq;
				force += ( FParams.iterm * pterm * dist + FParams.vterm * ( FPnts.bufF3(FVEVAL)[ j ] - FPnts.bufF3(FVEVAL)[i] )) * (FParams.psmoothradius-dsq) / (FPnts.bufF(FPRESS)[i] * FPnts.bufF(FPRESS)[ j ] );
			}	
		}
		//--- not used (function call is slow, uses too many registers)
		// force += contributeForce ( i, FPnts.bufF3(FPOS)[ i ], FPnts.bufF3(FVEVAL)[ i ], 1/FPnts.bufF(FPRESS)[ i ], (FPnts.bufF(FPRESS)[i] - FParams.prest_dens ) * FParams.pintstiff, gc + FParams.gridAdj[c] );
	}
	if ( isnan(force.x) || isnan(force.y) || isnan(force.z) ) force = make_float3(0,0,0);

	FPnts.bufF3(FFORCE)[ i ] = force;				

}

// gravity only
extern "C" __global__ void forceGravity ( int numPnts, float factor )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= numPnts ) return;

	FPnts.bufF3(FFORCE)[i] += FParams.gravity * factor / FParams.pmass;			// gravity added here (before boundary conditions)
}


#define COLORA(r,g,b,a)	( (uint(a*255.0f)<<24) | (uint(b*255.0f)<<16) | (uint(g*255.0f)<<8) | uint(r*255.0f) )
 
extern "C" __global__ void advanceParticles ( float time, float dt, float ss, int numPnts )
{		
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= numPnts  ) return;
		
	// Get particle vars
	register float3 accel, norm;	
	register float3 pos = FPnts.bufF3(FPOS)[i];
	register float3 vel = FPnts.bufF3(FVEL)[i];
	register float3 veval = FPnts.bufF3(FVEVAL)[i];	
	register float diff, adj, speed;

	accel = FPnts.bufF3(FFORCE)[i] * FParams.pmass;
		
	// Boundaries
	// Y-axis	
	diff = FParams.pradius - (pos.y - (FParams.bound_min.y + (pos.x-FParams.bound_min.x)*FParams.bound_slope )) * ss;
	if ( diff > EPSILON ) {
		norm = make_float3( -FParams.bound_slope, 1.0 - FParams.bound_slope, 0);
		adj = FParams.bound_stiff * diff - FParams.bound_damp * dot(norm, veval );		
		norm *= adj; accel += norm - veval * FParams.bound_friction;
	}

	diff = FParams.pradius - ( FParams.bound_max.y - pos.y )*ss;
	if ( diff > EPSILON ) {
		norm = make_float3(0, -1, 0);
		adj = FParams.bound_stiff * diff - FParams.bound_damp * dot(norm, veval );
		norm *= adj; accel += norm - veval * FParams.bound_friction;
	}

	// X-axis
	//float wall = (sin(time*FParams.bound_wall_freq + pos.z/400.f)*0.5+0.5) * FParams.bound_wall_force;
	diff = FParams.pradius - (pos.x - FParams.bound_min.x )*ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 1, 0, 0);
		adj = FParams.bound_stiff * diff - FParams.bound_damp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
	diff = FParams.pradius - ( FParams.bound_max.x - pos.x)*ss;
	if ( diff > EPSILON ) {
		norm = make_float3(-1, 0, 0);
		adj = FParams.bound_stiff * diff - FParams.bound_damp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	// Z-axis
	diff = FParams.pradius - (pos.z - FParams.bound_min.z ) * ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 0, 0, 1 );
		adj = FParams.bound_stiff * diff - FParams.bound_damp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
	diff = FParams.pradius - ( FParams.bound_max.z - pos.z )*ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 0, 0, -1 );
		adj = FParams.bound_stiff * diff - FParams.bound_damp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	// Accel Limit
	speed = accel.x*accel.x + accel.y*accel.y + accel.z*accel.z;
	if ( speed > FParams.AL2 ) {
		accel *= FParams.AL / sqrt(speed);
	}
	// Velocity Limit	
	speed = vel.x*vel.x + vel.y*vel.y + vel.z*vel.z;
	if ( speed > FParams.VL2 ) {		
		vel *= FParams.VL / sqrt(speed);
	}

	// Velocity Verlet Integration
	float3 vhalf = vel + accel*dt*0.5;					// v(t+1/2) = v(t) + 1/2*a(t)*dt				
	FPnts.bufF3(FPOS)[i] += vhalf * (dt/ss);			// p(t+1) = p(t) + v(t+1/2) dt		
	FPnts.bufF3(FVEL)[i] = vhalf;
	FPnts.bufF3(FVEVAL)[i] = (vel + vhalf) * 0.5;		// v(t+1) = [v(t-1/2) + v(t+1/2)] * 0.5
	FPnts.bufF3(FFORCE)[i] = make_float3(0,0,0);

	//-- color from velocity
	/*float vlen = length(vel)/0.18f; 
	float4 vclr = make_float4(0,0,1,1) + min(1.0f, vlen*vlen*vlen)*(make_float4(0,1,1,1)-make_float4(0,0,1,1));
	FPnts.bufI(FCLR)[i] = VECCLR(vclr); */
}

extern "C" __global__ void prefixFixup(uint *input, uint *aux, int len)
{
	unsigned int t = threadIdx.x;
	unsigned int start = t + 2 * blockIdx.x * SCAN_BLOCKSIZE;
	if (start < len)					input[start] += aux[blockIdx.x];
	if (start + SCAN_BLOCKSIZE < len)   input[start + SCAN_BLOCKSIZE] += aux[blockIdx.x];
}

extern "C" __global__ void prefixSum(uint* input, uint* output, uint* aux, int len, int zeroff)
{
	__shared__ uint scan_array[SCAN_BLOCKSIZE << 1];
	unsigned int t1 = threadIdx.x + 2 * blockIdx.x * SCAN_BLOCKSIZE;
	unsigned int t2 = t1 + SCAN_BLOCKSIZE;

	// Pre-load into shared memory
	scan_array[threadIdx.x] = (t1<len) ? input[t1] : 0.0f;
	scan_array[threadIdx.x + SCAN_BLOCKSIZE] = (t2<len) ? input[t2] : 0.0f;
	__syncthreads();

	// Reduction
	int stride;
	for (stride = 1; stride <= SCAN_BLOCKSIZE; stride <<= 1) {
		int index = (threadIdx.x + 1) * stride * 2 - 1;
		if (index < 2 * SCAN_BLOCKSIZE)
			scan_array[index] += scan_array[index - stride];
		__syncthreads();
	}

	// Post reduction
	for (stride = SCAN_BLOCKSIZE >> 1; stride > 0; stride >>= 1) {
		int index = (threadIdx.x + 1) * stride * 2 - 1;
		if (index + stride < 2 * SCAN_BLOCKSIZE)
			scan_array[index + stride] += scan_array[index];
		__syncthreads();
	}
	__syncthreads();

	// Output values & aux
	if (t1 + zeroff < len)	output[t1 + zeroff] = scan_array[threadIdx.x];
	if (t2 + zeroff < len)	output[t2 + zeroff] = (threadIdx.x == SCAN_BLOCKSIZE - 1 && zeroff) ? 0 : scan_array[threadIdx.x + SCAN_BLOCKSIZE];
	if (threadIdx.x == 0) {
		if (zeroff) output[0] = 0;
		if (aux) aux[blockIdx.x] = scan_array[2 * SCAN_BLOCKSIZE - 1];
	}
}
